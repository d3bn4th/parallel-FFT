#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <chrono> 


using namespace std::chrono;

//#include <cv.h>
//#include <highgui.h>

using namespace std;
using namespace cv;

typedef complex<float> base;
typedef float2 Complex_my;

template <typename T>
ostream &operator<<(ostream &o, vector<T> v)
{
    if (v.size() > 0)
        o << v[0];
    for (unsigned i = 1; i < v.size(); i++)
        o << " " << v[i];
    return o << endl;
}
static __device__ __host__ inline Complex_my Add(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x + B.x;
    C.y = A.y + B.y;
    return C;
}

/**
 *  Inverse of Complex_my Number
 */
static __device__ __host__ inline Complex_my Inverse(Complex_my A)
{
    Complex_my C;
    C.x = -A.x;
    C.y = -A.y;
    return C;
}

/**
 *  Multipication of Complex_my Numbers
 */
static __device__ __host__ inline Complex_my Multiply(Complex_my A, Complex_my B)
{
    Complex_my C;
    C.x = A.x * B.x - A.y * B.y;
    C.y = A.y * B.x + A.x * B.y;
    return C;
}

/**
* Parallel Functions for performing various tasks
*/

/**
*  Dividing by constant for inverse fft transform
*/
__global__ void inplace_divide_invert(Complex_my *A, int n, int threads)
{
    int i = blockIdx.x * threads + threadIdx.x;
    if (i < n)
    {
        // printf("in divide");
        A[i].x /= n;
        A[i].y /= n;
    }
    else
    {
        // printf("else in divide");
        // printf("i=%d, n=%d", i, n);
    }
}

/**
* Reorders array by bit-reversing the indexes.
*/
__global__ void bitrev_reorder(Complex_my *__restrict__ r, Complex_my *__restrict__ d, int s, size_t nthr, int n)
{
    int id = blockIdx.x * nthr + threadIdx.x;
    //r[id].x = -1;
    if (id < n and __brev(id) >> (32 - s) < n)
        r[__brev(id) >> (32 - s)] = d[id];
}

/**
* Inner part of the for loop
*/
__device__ void inplace_fft_inner(Complex_my *__restrict__ A, int i, int j, int len, int n, bool invert)
{
    if (i + j + len / 2 < n and j < len / 2)
    {
        Complex_my u, v;

        float angle = (2 * M_PI * j) / (len * (invert ? -1.0 : 1.0));
        v.x = cos(angle);
        v.y = sin(angle);

        u = A[i + j];
        v = Multiply(A[i + j + len / 2], v);
        // printf("i:%d j:%d u_x:%f u_y:%f    v_x:%f v_y:%f\n", i, j, u.x, u.y, v.x, v.y);
        A[i + j] = Add(u, v);
        A[i + j + len / 2] = Add(u, Inverse(v));
    }
}

/**
* FFT if number of threads are sufficient.
*/
__global__ void inplace_fft(Complex_my *__restrict__ A, int i, int len, int n, int threads, bool invert)
{
    int j = blockIdx.x * threads + threadIdx.x;
    inplace_fft_inner(A, i, j, len, n, invert);
}

/**
* FFt if number of threads are not sufficient.
*/
__global__ void inplace_fft_outer(Complex_my *__restrict__ A, int len, int n, int threads, bool invert)
{
    int i = (blockIdx.x * threads + threadIdx.x);
    for (int j = 0; j < len / 2; j++)
    {
        inplace_fft_inner(A, i, j, len, n, invert);
    }
}

/**
* parallel FFT transform and inverse transform
* Arguments vector of complex numbers, invert, balance, number of threads
* Perform inplace transform
*/
void fft(vector<base> &a, bool invert, int balance = 10, int threads = 32)
{
    // Creating array from vector
    int n = (int)a.size();
    int data_size = n * sizeof(Complex_my);
    Complex_my *data_array = (Complex_my *)malloc(data_size);
    for (int i = 0; i < n; i++)
    {
        data_array[i].x = a[i].real();
        data_array[i].y = a[i].imag();
    }
    
    // Copying data to GPU
    Complex_my *A, *dn;
    hipMalloc((void **)&A, data_size);
    hipMalloc((void **)&dn, data_size);
    hipMemcpy(dn, data_array, data_size, hipMemcpyHostToDevice);
    // Bit reversal reordering
    int s = log2(n);

    bitrev_reorder<<<ceil(float(n) / threads), threads>>>(A, dn, s, threads, n);

    
    // Synchronize
    hipDeviceSynchronize();
    // Iterative FFT with loop parallelism balancing
    for (int len = 2; len <= n; len <<= 1)
    {
        if (n / len > balance)
        {

            inplace_fft_outer<<<ceil((float)n / threads), threads>>>(A, len, n, threads, invert);
        }
        else
        {
            for (int i = 0; i < n; i += len)
            {
                float repeats = len / 2;
                inplace_fft<<<ceil(repeats / threads), threads>>>(A, i, len, n, threads, invert);
            }
        }
    }
    
    if (invert)
        inplace_divide_invert<<<ceil(n * 1.00 / threads), threads>>>(A, n, threads);

    // Copy data from GPU
    Complex_my *result;
    result = (Complex_my *)malloc(data_size);
    hipMemcpy(result, A, data_size, hipMemcpyDeviceToHost);
    
    // Saving data to vector<complex> in input.
    for (int i = 0; i < n; i++)
    {
        a[i] = base(result[i].x, result[i].y);
    }
    // Free the memory blocks
    free(data_array);
    hipFree(A);
    hipFree(dn);
    return;
}

/**
* Performs 2D FFT 
* takes vector of complex vectors, invert and verbose as argument
* performs inplace FFT transform on input vector
*/
void fft2D(vector<vector<base>> &a, bool invert, int verbose = 0)
{
    auto matrix = a;
    // Transform the rows
    if (verbose > 0)
        cout << "Transforming Rows" << endl;

    for (auto i = 0; i < matrix.size(); i++)
    {
        //cout<<i<<endl;
        fft(matrix[i], invert);
    }

    // preparing for transforming columns

    if (verbose > 0)
        cout << "Converting Rows to Columns" << endl;

    a = matrix;
    matrix.resize(a[0].size());
    for (int i = 0; i < matrix.size(); i++)
        matrix[i].resize(a.size());

    // Transposing matrix
    for (int i = 0; i < a.size(); i++)
    {
        for (int j = 0; j < a[0].size(); j++)
        {
            matrix[j][i] = a[i][j];
        }
    }
    if (verbose > 0)
        cout << "Transforming Columns" << endl;

    // Transform the columns
    for (auto i = 0; i < matrix.size(); i++)
        fft(matrix[i], invert);

    if (verbose > 0)
        cout << "Storing the result" << endl;

    // Storing the result after transposing
    // [j][i] is getting value of [i][j]
    for (int i = 0; i < a.size(); i++)
    {
        for (int j = 0; j < a[0].size(); j++)
        {
            a[j][i] = matrix[i][j];
        }
    }
}

/**
* Function to multiply two polynomial
* takes two polynomials represented as vectors as input
* return the product of two vectors
*/
vector<int> mult(vector<int> a, vector<int> b, int balance, int threads)
{
    // Creating complex vector from input vectors
    vector<base> fa(a.begin(), a.end()), fb(b.begin(), b.end());

    // Padding with zero to make their size equal to power of 2
    size_t n = 1;
    while (n < max(a.size(), b.size()))
        n <<= 1;
    n <<= 1;

    fa.resize(n), fb.resize(n);

    // Transforming both a and b
    // Converting to points form
    fft(fa, false, balance, threads), fft(fb, false, balance, threads);

    // performing point wise multipication of points
    for (size_t i = 0; i < n; ++i)
        fa[i] *= fb[i];

    // Performing Inverse transform
    fft(fa, true, balance, threads);

    // Saving the real part as it will be the result
    vector<int> res;
    res.resize(n);
    for (size_t i = 0; i < n; ++i)
        res[i] = int(fa[i].real() + 0.5);

    return res;
}

class FFT
{
public:
    /**
     * parallel FFT transform and inverse transform
     * Arguments vector of complex numbers, invert, balance, number of threads
     * Perform inplace transform
     */
    void fft(vector<base> &a, bool invert)
    {
        // Performing Bit reversal ordering
        int n = (int)a.size();

        for (int i = 1, j = 0; i < n; ++i)
        {
            int bit = n >> 1;
            for (; j >= bit; bit >>= 1)
                j -= bit;
            j += bit;
            if (i < j)
                swap(a[i], a[j]);
        }

        // Iteratinve FFT
        // This part of FFT is parallelizable
        for (int len = 2; len <= n; len <<= 1)
        {
            double ang = 2 * M_PI / len * (invert ? 1 : -1);
            base wlen(cos(ang), sin(ang));
            for (int i = 0; i < n; i += len)
            {
                base w(1);
                for (int j = 0; j < len / 2; ++j)
                {
                    base u = a[i + j], v = a[i + j + len / 2] * w;
                    a[i + j] = u + v;
                    a[i + j + len / 2] = u - v;
                    w *= wlen;
                }
            }
        }

        if (invert)
            for (int i = 0; i < n; ++i)
                a[i] /= n;
        return;
    }

    /**
     * Performs 2D FFT 
     * takes vector of complex vectors, invert and verbose as argument
     * performs inplace FFT transform on input vector
     */
    void fft2D(vector<vector<base>> &a, bool invert, int verbose = 0)
    {
        auto matrix = a;
        // Transform the rows
        if (verbose > 0)
            cout << "Transforming Rows" << endl;

        for (auto i = 0; i < matrix.size(); i++)
        {
            //cout<<i<<endl;
            fft(matrix[i], invert);
        }

        // preparing for transforming columns

        if (verbose > 0)
            cout << "Converting Rows to Columns" << endl;

        a = matrix;
        matrix.resize(a[0].size());
        for (int i = 0; i < matrix.size(); i++)
            matrix[i].resize(a.size());

        // Transposing matrix
        for (int i = 0; i < a.size(); i++)
        {
            for (int j = 0; j < a[0].size(); j++)
            {
                matrix[j][i] = a[i][j];
            }
        }
        if (verbose > 0)
            cout << "Transforming Columns" << endl;

        // Transform the columns
        for (auto i = 0; i < matrix.size(); i++)
            fft(matrix[i], invert);

        if (verbose > 0)
            cout << "Storing the result" << endl;

        // Storing the result after transposing
        // [j][i] is getting value of [i][j]
        for (int i = 0; i < a.size(); i++)
        {
            for (int j = 0; j < a[0].size(); j++)
            {
                a[j][i] = matrix[i][j];
            }
        }
    }

    /**
     * Function to multiply two polynomial
     * takes two polynomials represented as vectors as input
     * return the product of two vectors
     */
    vector<int> mult(vector<int> a, vector<int> b)
    {
        // Creating complex vector from input vectors
        vector<base> fa(a.begin(), a.end()), fb(b.begin(), b.end());

        // Padding with zero to make their size equal to power of 2
        size_t n = 1;
        while (n < max(a.size(), b.size()))
            n <<= 1;
        n <<= 1;

        fa.resize(n), fb.resize(n);

        // Transforming both a and b
        // Converting to points form
        fft(fa, false), fft(fb, false);

        // performing point wise multipication of points
        for (size_t i = 0; i < n; ++i)
            fa[i] *= fb[i];

        // Performing Inverse transform
        fft(fa, true);

        // Saving the real part as it will be the result
        vector<int> res;
        res.resize(n);
        for (size_t i = 0; i < n; ++i)
            res[i] = int(fa[i].real() + 0.5);

        return res;
    }

    /**
     * Function to perform jpeg compression on image
     * takes image, threshold, verbose as input
     * image is represented as vector<vector>
     * perform inplace compression on the input
     */
    void compress_image(vector<vector<uint8_t>> &image, double threshold, int verbose = 1)
    {
        //Convert image to complex type

        vector<vector<base>> complex_image(image.size(), vector<base>(image[0].size()));
        for (auto i = 0; i < image.size(); i++)
        {
            for (auto j = 0; j < image[0].size(); j++)
            {
                complex_image[i][j] = image[i][j];
            }
        }
        if (verbose == 1)
        {
            cout << "input Image" << endl;
            //cout << image;
            cout << endl
                 << endl;
        }
        if (verbose > 1)
        {
            cout << "Complex Image" << endl;
            cout << complex_image;
            cout << endl
                 << endl;
        }

        //Perform 2D fft on image

        fft2D(complex_image, false, verbose);

        if (verbose == 1)
        {
            cout << "Performing FFT on Image" << endl;
            ///cout << complex_image;
            cout << endl
                 << endl;
        }

        //Threshold the fft

        // for (int i = 0; i < image_M.rows; ++i)
        //     for (int j = 0; j < image_M.cols; ++j)
        //         image_M.at<uint8_t>(i, j) = image[i][j];

        double maximum_value = 0.0;
        for (int i = 0; i < complex_image.size(); i++)
        {
            for (int j = 0; j < complex_image[0].size(); j++)
            {
                maximum_value = max(maximum_value, abs(complex_image[i][j]));
            }
        }
        threshold *= maximum_value;
        cout << "threshold :" << threshold << endl;
        int count = 0;

        // Setting values less than threshold to zero
        // This step is responsible for compression
        for (int i = 0; i < complex_image.size(); i++)
        {
            for (int j = 0; j < complex_image[0].size(); j++)
            {
                if (abs(complex_image[i][j]) < threshold)
                {
                    count++;
                    complex_image[i][j] = 0;
                }
            }
        }
        cout << count << endl;
        if (verbose > 1)
        {
            cout << "Thresholded Image" << endl;
            //cout << complex_image;
            cout << endl
                 << endl;
        }

        // Perform inverse FFT
        fft2D(complex_image, true, verbose);
        if (verbose > 1)
        {
            cout << "Inverted Image" << endl;
            //cout << complex_image;
            cout << endl
                 << endl;
        }
        //Convert to uint8 format
        // We will consider only the real part of the image
        for (int i = 0; i < complex_image.size(); i++)
        {
            for (int j = 0; j < complex_image[0].size(); j++)
            {
                image[i][j] = uint8_t(complex_image[i][j].real() + 0.5);
            }
        }
        if (verbose > 0)
        {
            cout << "Compressed Image" << endl;
            //cout << image;
        }
    }
};


#define N 1000
#define BALANCE 2

int main()
{
    vector<int> a = {1,1};
    vector<int> b = {1,2,3};
    auto multiplier = FFT();
    cout<<"A = "<<a;
    cout<<"B = "<<b;
    cout<<"A * B = "<<multiplier.mult(a, b)<<endl;
    /*
    std::vector<int> fa(N);
    std::generate(fa.begin(), fa.end(), std::rand);
    std::vector<int> fb(N);
    std::generate(fb.begin(), fb.end(), std::rand);
    freopen("out.txt", "w", stdout);
    for(int threads = 1; threads <= 1024; threads*=2){
	cerr << "For threads= " << threads << endl;
        /// For Parallel
        auto start = high_resolution_clock::now(); 

        auto result_parallel = mult(fa, fb, BALANCE, threads);

        auto stop = high_resolution_clock::now(); 
        auto duration = duration_cast<microseconds>(stop - start); 
      
        cout << threads << " " << duration.count();


        /// For Sequential
        auto multiplier = FFT();
        
        start = high_resolution_clock::now(); 
        auto result_sequential = multiplier.mult(fa, fb);

        stop = high_resolution_clock::now(); 
        duration = duration_cast<microseconds>(stop - start); 

        cout << " " << duration.count();

        cout << " " << (result_parallel == result_sequential) << endl;
        cout << endl;
    }
    */
    return 0;
}
